#include <hip/hip_runtime.h>
#include <bits/stdc++.h>

using namespace std;

void fill_matrix_random(int *mat, int rows, int cols){
  for (int i = 0; i < rows; i++){
    for (int j = 0; j < cols; j++){
      mat[i * cols + j] = rand() % 99;
    }
  }
}

bool check_matrix(int *A, int *B, int rows, int cols){
  for (int i = 0; i < rows; i++){
    for (int j = 0; j < cols; j++){
      if (A[i * cols + j] != B[i * cols +j])
        return false;
    }
  }
  return true;
}

void print_matrix(int *mat, int rows, int cols){
  cout << "------------" << endl;
  for (int i = 0; i < rows; i++){
    for (int j = 0; j < cols; j++){
      cout << mat[i * cols + j] << " ";
    }
    cout << endl;
  }
  cout << "------------" << endl;
}

void mat_mul_seq(int *m_A, int *m_B, int *m_C, int A_rows, int A_cols, int B_rows, int B_cols){
  int sum;
  for(int i = 0; i < A_rows; i++){
    for (int j = 0; j < B_cols; j++){
      sum = 0;
      for (int k = 0; k < A_cols; k++){
        sum += m_A[i * A_cols + k] * m_B[k * B_cols + j];
      }
      m_C[i * B_cols + j] = sum;
    }
  }
}
__global__ void mat_mul_kernel(int *m_A, int *m_B, int *m_C, int A_rows, int A_cols, int B_rows, int B_cols){
  int sum = 0;
  int row = threadIdx.y + blockIdx.y * blockDim.y;
  int col = threadIdx.x + blockIdx.x * blockDim.x;
  if(row < A_rows && col < B_cols){
    for(int i = 0; i < A_cols; i ++){
      sum += m_A[row * A_cols + i] * m_B[i * B_cols + col];
    }
    m_C[row * B_cols + col] = sum;
  }
}

void mat_mul_con(int *m_A, int *m_B, int *m_C, int A_rows, int A_cols, int B_rows, int B_cols){
    int A_size = A_rows * A_cols * sizeof(int);
    int B_size = B_rows * B_cols * sizeof(int);
    int C_size = A_rows * B_cols * sizeof(int);

    int *d_A, *d_B, *d_C;
    //1. Allocate memory for d_A, etc. on the device (cudaMalloc)
    hipMalloc(&d_A, A_size);
    hipMalloc(&d_B, B_size);
    hipMalloc(&d_C, C_size);
    //2. Copy Data from host to d_A, etc. (cudaMemcpy)
    hipMemcpy(d_A, m_A, A_size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, m_B, B_size, hipMemcpyHostToDevice);
    //3. Kernel Launch Code
    dim3 dimGrid(ceil(max(A_rows, B_rows)/32.0), ceil(max(A_cols, B_cols)/32.0), 1);
    dim3 dimBlock(32, 32, 1);
    mat_mul_kernel<<<dimGrid, dimBlock>>> (d_A, d_B, d_C, A_rows, A_cols, B_rows, B_cols);
    hipDeviceSynchronize();
    //4. Copy d_C to C from device, free device memory (cusdaFree), sync if neccessary
    hipMemcpy (m_C, d_C, C_size, hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main(int argc, char **argv){
  if (argc < 5){
    cout << "Usage: ./mul max_number step offset_A, offset_B" << endl;
    return 0;
  }
  const int max_number = atoi(argv[1]),
  step = atoi(argv[2]),
  offset_A = atoi(argv[3]),
  offset_B = atoi(argv[4]);
  srand (time(NULL));

  ofstream x("x.mio"),
  y_seq("y_seq.mio"),
  y_con("y_con.mio");

  clock_t begin, end;
  double elapsed_secs;

  for (int i = step; i < max_number; i += step){
    int *A, *B, *C, *D;
    A = (int*) malloc((i + offset_A) * i * sizeof(int));
    B = (int*) malloc((i + offset_B) * i * sizeof(int));
    C = (int*) malloc((i + offset_A) * (i + offset_B) * sizeof(int));
    D = (int*) malloc((i + offset_A) * (i + offset_B) * sizeof(int));

    x << i << endl;

    fill_matrix_random(A, i + offset_A, i);
    fill_matrix_random(B, i, i + offset_B);

    begin = clock();
    mat_mul_seq(A, B, C, i + offset_A, i, i, i + offset_B);
    end = clock();
    elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
    y_seq << elapsed_secs << endl;

    begin = clock();
    mat_mul_con(A, B, D, i + offset_A, i, i, i + offset_B);
    end = clock();
    elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
    y_con << elapsed_secs << endl;

    if (check_matrix(C, D, i + offset_A, i + offset_B))
      cout << "All good" << endl;
    else
      cout << "Something Went Wrong" << endl;

    free(A);
    free(B);
    free(C);
    free(D);
  }

  return 0;
}
